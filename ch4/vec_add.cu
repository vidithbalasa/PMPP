
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <numeric>

__global__ void vecAddKernel(int* a, int* b, int* c, int n) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < n) {
        c[x] = a[x] + b[x];
    }
}

int main() {
    int n = 100000000;
    int size = n * sizeof(int);
    std::vector<int> a(n);
    std::vector<int> b(n);
    std::vector<int> c(n);
    // int c[n];

    std::iota(a.begin(), a.end(), 0);
    std::iota(b.begin(), b.end(), 0);

    int *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b.data(), size, hipMemcpyHostToDevice);

    vecAddKernel<<<(size + 256 - 1 / 256.0), 256>>>(d_A, d_B, d_C, n);

    hipMemcpy(c.data(), d_C, size, hipMemcpyDeviceToHost);

    /*
    for (int i=0; i<n; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;
    */
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
